#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <time.h>
#include <math.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

#define THREADS 512 

/**
 * Function on CPU code
 */
void SumOnCPU (int* sum, const int* datas, const int numElements)
{
	int i;
	*sum = 0;
	for (i=0; i<numElements; ++i)
		*sum += *(datas+i);
}

void UnrolledSumOnCPU (int* sum, const int* datas, const int numElements)
{
	int i;
	*sum = 0;
	for (i=0; i<numElements; i+=2)
		*sum += *(datas+i);
	for (i=1; i<numElements; i+=2)
		*sum += *(datas+i);
}

/**
 * CUDA Kernel Device code
 */

// global atomicAdd
__global__ void SumAtomic(int* sum, const int* datas, const int numElements)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;
	if (id >= numElements) return;
	atomicAdd(sum, datas[id]);
}

// shared atomicAdd
__global__ void SumReductionAtomic(int* sum, const int* datas, const int numElements)
{
	__shared__ int sh_sum;
	int id = blockDim.x * blockIdx.x + threadIdx.x;
	if (id >= numElements) return;
	
	if (threadIdx.x==0)
		sh_sum = 0;
	__syncthreads();

	atomicAdd(&sh_sum, datas[id]);
	__syncthreads();

	if (threadIdx.x == 0) atomicAdd(sum, sh_sum);
}

// binary sum reduction 
__global__ void SumReductionBinary(int* sum, const int* datas, const int numElements)
{
	__shared__ int sh_arr[THREADS]; // size: nThreads
	int id = blockDim.x * blockIdx.x + threadIdx.x;

	if (id >= numElements) {
		sh_arr[threadIdx.x] = 0;
	} else {
		sh_arr[threadIdx.x] = datas[id];
	}
	__syncthreads();

	int offset;
	for (offset=blockDim.x>>1; offset>0; offset>>=1) {
		if (threadIdx.x < offset)
			sh_arr[threadIdx.x] += sh_arr[threadIdx.x + offset];
		__syncthreads();
	}

	if (threadIdx.x == 0)
		atomicAdd(sum, sh_arr[0]);
}

/**
  * shuffle
  */

// shuffle sum reduce function with __shfl_down()
__device__ int reduceSumWarp(int val)
{
	for (int offset = warpSize/2; offset > 0; offset /= 2)
		val += __shfl_down(val, offset);
	return val;
}
// if all threads in the warp need the result, you have to using __shfl_xor()
__device__ int reduceInWarp(int val)
{
	for (int mask=warpSize/2; mask > 0; mask/=2) {
		val += __shfl_xor(val, mask);
	}
	return val;
}

__global__ void SumReductionShuffleAtom(int* sum, const int* datas, const int numElements)
{
	int id = blockDim.x * blockIdx.x + threadIdx.x;
	int idInWarp = threadIdx.x % warpSize;

	if (id < numElements) {
		int val = datas[id];
//		val = reduceInWarp(val);
		val = reduceSumWarp(val);
		if (idInWarp == 0)
			atomicAdd (sum, val);
	}
}

__global__ void SumReductionShuffleShared(int* sum, const int* datas, const int numElements)
{
	__shared__ int s_sum;
	int id = blockDim.x * blockIdx.x + threadIdx.x;
	int idInWarp = threadIdx.x % warpSize;
	if (threadIdx.x == 0)
		s_sum = 0;

	if (id < numElements) {
		int val = datas[id];
		val = reduceSumWarp(val);
		if (idInWarp == 0)
			atomicAdd(&s_sum, val);
		__syncthreads();
		if (threadIdx.x == 0)
			atomicAdd (sum, s_sum);
	}
}

__device__ int blockReduceSum(int val) {
	static __shared__ float shared[32];
	int lane = threadIdx.x % warpSize;	// thread index within the warp
	int wid  = threadIdx.x / warpSize;	// warp ID

	// warp reduction (only the threads with 0 index within the warp has warp reduction result)
	val = reduceSumWarp(val);
	if (lane == 0) shared[wid] = val;
	__syncthreads();

	// there will be at most 1024 threads within a block and at most 1024 blocks within a grid.
	val = (threadIdx.x < blockDim.x / warpSize) ? shared[lane] : 0;

	if (wid == 0) val = reduceSumWarp(val);

	return val;
}

__global__ void SumReductionShuffleQ(int* sum, const int* datas, const int numElements)
{
	int tSum = 0;
	for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < numElements; i += blockDim.x * gridDim.x)
		tSum += datas[i];
	tSum = blockReduceSum(tSum);
	if (threadIdx.x == 0)
		atomicAdd(sum, tSum);
}

float getduration (const struct timeval begin, const struct timeval finish)
{
	return ((finish.tv_sec-begin.tv_sec) * 1000.0
			+(double)(finish.tv_usec-begin.tv_usec)/1000.0);
}

/**
 * Host main routine
 */
int main(int argc, char **argv)
{
	if (argc < 3) {
		printf("ERROR: have to insert arguments\n");
		printf("usage: %s {# of elements} {sum reduction method}\n", argv[0]);
		printf(" - sum reduction methods are\n\
  * cpu : run sum reduction by for loop in cpu thread\n\
  * cpu_ur : loop unrolling with cpu\n\
  * g_atom : run sum reduction by global atomicAdd in gpu kernel\n\
  * s_atom : shared atomicAdd with gpu\n\
  * binary : binary sum reduction with gpu\n\
  * shfl_a : shuffle + global atomicAdd with gpu\n\
  * shfl_s : shuffle + shared atomicAdd with gpu\n\
  * shfl_q : quantitative reduction with shuffle\n");

		return -1;
	}

	struct timeval begin, finish;
	int numElements = atoi(argv[1]);
	int *datas, *d_datas;
	int sum, *d_sum;

	// init value
	datas = (int*) malloc(sizeof(int) * numElements);
	for (int i=0; i<numElements; ++i)
		datas[i] = 1;

	// Device Memory Allocate
	hipMalloc((void **)&d_datas, sizeof(int) * numElements);
	hipMalloc((void **)&d_sum, sizeof(int));

	// Copy the Host Value to Device
	hipMemcpy(d_datas, datas, sizeof(int) * numElements, hipMemcpyHostToDevice);
	hipMemset(d_sum, 0, sizeof(int));

	printf("\nSummation %d elements\n", numElements);

	// set CUDA block/thread
	int threads = THREADS;
	int blocks = numElements % threads ? (numElements / threads) + 1 : numElements / threads;

	// run sum reduction & print result
	gettimeofday(&begin, NULL);
	if (!strcmp(argv[2], "cpu"))
		SumOnCPU(&sum, datas, numElements);
	else if (!strcmp(argv[2], "cpu_ur"))
		UnrolledSumOnCPU(&sum, datas, numElements);
	else if (!strcmp(argv[2], "g_atom")) {
		SumAtomic<<<blocks, threads>>>(d_sum, d_datas, numElements);
		hipMemcpy(&sum, d_sum, sizeof(int), hipMemcpyDeviceToHost);
	} else if (!strcmp(argv[2], "s_atom")) {
		SumReductionAtomic<<<blocks, threads>>>(d_sum, d_datas, numElements);
		hipMemcpy(&sum, d_sum, sizeof(int), hipMemcpyDeviceToHost);
	} else if (!strcmp(argv[2], "binary")) {
		SumReductionBinary<<<blocks, threads>>>(d_sum, d_datas, numElements);
		hipMemcpy(&sum, d_sum, sizeof(int), hipMemcpyDeviceToHost);
	} else if (!strcmp(argv[2], "shfl_a")) {
		SumReductionShuffleAtom<<<blocks, threads>>>(d_sum, d_datas, numElements);
		hipMemcpy(&sum, d_sum, sizeof(int), hipMemcpyDeviceToHost);
	} else if (!strcmp(argv[2], "shfl_s")) {
		SumReductionShuffleShared<<<blocks, threads>>>(d_sum, d_datas, numElements);
		hipMemcpy(&sum, d_sum, sizeof(int), hipMemcpyDeviceToHost);
	} else if (!strcmp(argv[2], "shfl_q")) {
		SumReductionShuffleQ<<<1024, 1024>>>(d_sum, d_datas, numElements);
		hipMemcpy(&sum, d_sum, sizeof(int), hipMemcpyDeviceToHost);
	} else {
		printf("ERROR: you used wrong method\n");
		return -1;
	}
	gettimeofday(&finish, NULL);
	printf("** Used %s method\n", argv[2]);
	printf("  sum result: %d\n", sum);
	printf("  Sum Duration: %.3f(ms)\n", getduration(begin, finish));

	// Free device global memory
	hipFree(d_datas);
	hipFree(d_sum);

	// Free host memory
	free(datas);

	// Reset the device and exit
	hipDeviceReset();
	return 0;
}

